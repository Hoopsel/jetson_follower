#include "hip/hip_runtime.h"
#include "gpu_utils.cuh"

__global__ void gpu_rgb2hsv(uchar* data, int width, int height)
{
    int index_x = blockIdx.x*blockDim.x+threadIdx.x;
    int index_y = blockIdx.y*blockDim.y+threadIdx.y;
    int index = 3*(index_y*width + index_x);

    if (index_x<width && index_y<height)
    {
        float r = ((float)data[index+0])/255;
        float g = ((float)data[index+1])/255;
        float b = ((float)data[index+2])/255;

        float c_max = max(max(r, g), b);
        float c_min = min(min(r, g), b);
        float delta = c_max - c_min;

        data[index+2] = c_max*255;                  // v = c_max

        if (delta < 0.003)                          // d < 1/255
        {
            data[index+0] = 0;
            data[index+1] = 0;
        } else
        {
            if (c_max == r)
            {
                int h = (60*(g-b)/delta)/2;
                data[index+0] = (h < 0) ?  h+=180 : h;
            }
            else if (c_max == g)
            {
                int h = (120+60*(b-r)/delta)/2;
                data[index+0] = (h < 0) ?  h+=180 : h;
            }
            else if (c_max == b)
            {
                int h = (240+60*(r-g)/delta)/2;
                data[index+0] = (h < 0) ?  h+=180 : h;
            }

            data[index+1] = (delta * 255 / c_max);  // s = delta/c_max
        }
    }
}
__global__ void gpu_threshold(uchar* data_in, uchar* data_out, int width, int height, bool ch1_swap,
                          uchar ch1_min, uchar ch1_max, uchar ch2_min, uchar ch2_max, uchar ch3_min, uchar ch3_max)
{
    int index_x = blockIdx.x*blockDim.x+threadIdx.x;
    int index_y = blockIdx.y*blockDim.y+threadIdx.y;
    int index = (index_y*width + index_x);
    int index_3 = 3*index;

    if (index_x<width && index_y<height)
    {
        uchar ch1 = data_in[index_3+0];
        uchar ch2 = data_in[index_3+1];
        uchar ch3 = data_in[index_3+2];

        uchar out = 0;
        if (ch2 >= ch2_min && ch2 <= ch2_max &&
            ch3 >= ch3_min && ch3 <= ch3_max)
        {
            if (!ch1_swap && ch1 >= ch1_min && ch1 <= ch1_max)
                out = 255;

            if (ch1_swap && (ch1 < ch1_min || ch1 > ch1_max))
                out = 255;
        }

        data_out[index] = out;
    }
}

void convertAndThreshold(cv::Mat &mat, sensor_msgs::ImageConstPtr &image, bool h_swap,
                         int h_min, int h_max, int s_min, int s_max, int v_min, int v_max)
{
    int width = image->width;
    int height = image->height;

    assert(width == 640 && height == 480 &&
           !strcmp(image->encoding.c_str(),"rgb8"));

    uchar *d_data_in, *d_data_out;
    hipMalloc((void**)&d_data_in, 3*width*height*sizeof(uchar));
    hipMalloc((void**)&d_data_out, width*height*sizeof(uchar));
    hipMemcpy((void*)d_data_in, (void*)image->data.data(), 3*width*height*sizeof(uchar), hipMemcpyHostToDevice);

    dim3 blocks(width/32, height/32, 1);
    dim3 threads(32,32,1);
    gpu_rgb2hsv<<<blocks,threads>>>(d_data_in, width, height);
    gpu_threshold<<<blocks,threads>>>(d_data_in, d_data_out, width, height, h_swap, h_min, h_max, s_min, s_max, v_min, v_max);

    mat = cv::Mat(height, width, CV_8UC1);
    hipMemcpy((void*)mat.data, (void*)d_data_out, width*height*sizeof(uchar), hipMemcpyDeviceToHost);

    hipFree((void*)d_data_in);
    hipFree((void*)d_data_out);
}
